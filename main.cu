#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <opencv2/core.hpp>
#include <opencv2/core/matx.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <sstream>
#include <string>

#define CHANNELS 3 // RGB
#define M_PI 3.141592f

// Convert an index, image[channel][i][j] to flat[idx]
__device__ __host__ int getIdx(int width, int channel, int i, int j) {
    return j * width * CHANNELS + i * CHANNELS + channel;
}

__global__ void gaussianBlur(float* kernel, unsigned char* source, unsigned char* target, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int totalSize = width * height * CHANNELS;
        for (int ch = 0;ch < CHANNELS;ch++) {
            float weightedSum = 0;
            for (int i = -2;i <= 2;i++) {
                for (int j = -2;j <= 2;j++) {
                    int idx = getIdx(width, ch, x + i, y + j);
                    if (idx >= 0 && idx < totalSize) {
                        // Get the flat index + move indices from [-2,2] to [0,4] for the kernel
                        int kernelIdx = (i + 2) * 5 + (j + 2);

                        weightedSum += (int)source[idx] * kernel[kernelIdx];
                    }
                }
            }

            target[getIdx(width, ch, x, y)] = weightedSum;
        }
    }
}

__global__ void grayscale(unsigned char* imageData, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int channelAvg = 0;
        for (int ch = 0;ch < CHANNELS;ch++) {
            channelAvg += (int)imageData[getIdx(width, ch, x, y)];
        }
        channelAvg /= 3;
        for (int ch = 0;ch < CHANNELS;ch++) {
            imageData[getIdx(width, ch, x, y)] = channelAvg;
        }
    }
}

__global__ void generateMagnitudes(unsigned char* source, float* target, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int kernelX[3][3] = {
            {1, 0, -1},
            {2, 0, -2},
            {1, 0, -1},
        }, kernelY[3][3] = {
            { 1,  2,  1},
            { 0,  0,  0},
            {-1, -2, -1},
        };

        int totalSize = width * height * CHANNELS;
        for (int ch = 0;ch < CHANNELS;ch++) {
            int Gx = 0,
                Gy = 0;
            for (int i = -1;i <= 1;i++) {
                for (int j = -1;j <= 1;j++) {
                    int idx = getIdx(width, ch, x + i, y + j);
                    if (idx >= 0 && idx < totalSize) {
                        // Move indices from [-1,1] to [0,2] for the kernel
                        Gx += (int)source[idx] * kernelX[i + 1][j + 1];
                        Gy += (int)source[idx] * kernelY[i + 1][j + 1];
                    }
                }
            }

            float magnitude = sqrt((float)Gx * Gx + Gy * Gy);
            target[getIdx(width, ch, x, y)] = magnitude;
        }
    }
}

__global__ void hysteresis(unsigned char* source, unsigned char* target, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int totalSize = width * height * CHANNELS;
    for (int ch = 0;ch < CHANNELS;ch++) {
        bool strongEdge = false;

        for (int i = -1;i <= 1 && !strongEdge;i++) {
            for (int j = -1;j <= 1;j++) {
                int idx = getIdx(width, ch, x + i, y + j);
                if (idx >= 0 && idx < totalSize && source[idx] == 255) {
                    printf("%d\n", source[idx]);
                    strongEdge = true;
                    break;
                }
            }
        }

        target[getIdx(width, ch, x, y)] = strongEdge ? 255 : 0;
    }
}

__global__ void sobel(unsigned char* source, unsigned char* target, float* magnitudes, int width, int height, float lowThreshold, float highThreshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int kernelX[3][3] = {
            {1, 0, -1},
            {2, 0, -2},
            {1, 0, -1},
        }, kernelY[3][3] = {
            { 1,  2,  1},
            { 0,  0,  0},
            {-1, -2, -1},
        };

        int totalSize = width * height * CHANNELS;
        for (int ch = 0;ch < CHANNELS;ch++) {
            int Gx = 0,
                Gy = 0;
            for (int i = -1;i <= 1;i++) {
                for (int j = -1;j <= 1;j++) {
                    int idx = getIdx(width, ch, x + i, y + j);
                    if (idx >= 0 && idx < totalSize) {
                        //TODO don't generate twice
                        // Move indices from [-1,1] to [0,2] for the kernel
                        Gx += (int)source[idx] * kernelX[i + 1][j + 1];
                        Gy += (int)source[idx] * kernelY[i + 1][j + 1];
                    }
                }
            }

            // Intensity gradient
            float currentMagnitude = sqrtf(Gx * Gx + Gy * Gy);

            // Calculate the direction of the gradient
            float direction = atan2f(Gy, Gx);
            direction = direction * 180.0f / M_PI; // Convert to degrees
            if (direction < 0.0f) {
                direction += 180.0f;
            }

            // Non-maximum suppression
            // Compare with neighboring pixels
            float neighbor1, neighbor2;
            if (direction < 22.5f || direction >= 157.5f) { // North-South
                neighbor1 = y > 0 ? magnitudes[getIdx(width, ch, x, y - 1)] : 0.0f;
                neighbor2 = y < height - 1 ? magnitudes[getIdx(width, ch, x, y + 1)] : 0.0f;
            }
            else if (direction < 67.5f) { // North-East to South-West
                neighbor1 = x > 0 && y > 0 ? magnitudes[getIdx(width, ch, x - 1, y - 1)] : 0.0f;
                neighbor2 = x < width - 1 && y < height - 1 ? magnitudes[getIdx(width, ch, x + 1, y + 1)] : 0.0f;
            }
            else if (direction < 112.5f) { // East-West
                neighbor1 = x > 0 ? magnitudes[getIdx(width, ch, x - 1, y)] : 0.0f;
                neighbor2 = x < width - 1 ? magnitudes[getIdx(width, ch, x + 1, y)] : 0.0f;
            }
            else { // North-West to South-East
                neighbor1 = x > 0 && y < height - 1 ? magnitudes[getIdx(width, ch, x - 1, y + 1)] : 0.0f;
                neighbor2 = x < width - 1 && y > 0 ? magnitudes[getIdx(width, ch, x + 1, y - 1)] : 0.0f;
            }

            // Preserve the current pixel if it's the maximum
            int idx = getIdx(width, ch, x, y);
            if (currentMagnitude > neighbor1 && currentMagnitude > neighbor2) {
                // Double thresholding
                if (currentMagnitude > highThreshold) {
                    target[idx] = 255; // Strong edge pixel
                }
                else if (currentMagnitude > lowThreshold) {
                    target[idx] = 128; // Weak edge pixel
                }
                else {
                    target[idx] = 0; // Suppressed pixel
                }
            }
            else {
                target[idx] = 0;
            }
        }
    }
}

int main(int argc, char* argv[])
{
    std::string imagePath = "image.jpg";
    float lowThreshold = 0.2f,
        highThreshold = 0.6f;

    std::string propPath = "config.properties";
    std::ifstream propFile(propPath);
    if (propFile.is_open()) {
        std::string line;
        while (std::getline(propFile, line)) {
            size_t equalsPos = line.find('=');
            if (equalsPos != std::string::npos) {
                std::string key = line.substr(0, equalsPos);
                std::string value = line.substr(equalsPos + 1);

                if (key == "image") {
                    imagePath = value;
                }
                else if (key == "lowThreshold") {
                    lowThreshold = std::stof(value);
                }
                else if (key == "highThreshold") {
                    highThreshold = std::stof(value);
                }
            }
        }
        
    }
    else {
        printf("Could not open or find the properties file: %s\n", propPath);
    }

    cv::Mat image = cv::imread(imagePath, cv::IMREAD_ANYCOLOR);

    if (image.empty()) {
        printf("Could not open or find the image: %s\n", imagePath);
        return 1;
    }

    cv::namedWindow("Original", cv::WINDOW_AUTOSIZE);
    cv::imshow("Original", image);

    int width = image.cols,
        height = image.rows,
        channels = image.channels(); // Blue, green, red, etc.

    size_t imageDataSize = image.total() * image.channels();
    unsigned char* deviceImageData;
    hipMalloc(&deviceImageData, imageDataSize);
    hipMemcpy(deviceImageData, image.data, imageDataSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16, 1);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, 1);

    printf("blockSize: (%d,%d,%d)\n", blockSize.x, blockSize.y, blockSize.z);
    printf("gridSize: (%d,%d,%d)\n", gridSize.x, gridSize.y, gridSize.z);

    printf("Applying grayscale\n");
    grayscale << <gridSize, blockSize >> > (deviceImageData, width, height);

    printf("Applying Gaussian blur\n");
    float kernel[5][5]{},
        kernelSum = .0f,
        sigma = .75f;

    for (int x = -2;x <= 2;x++) {
        for (int y = -2;y <= 2;y++) {
            int i = x + 2,
                j = y + 2;
            kernel[i][j] = 1 / (2 * M_PI * sigma * sigma) * exp(-((float)x * x + y * y) / 2 * sigma * sigma);
            kernelSum += kernel[i][j];
        }
    }

    // Normalize
    for (int i = 0;i < 5;i++) {
        for (int j = 0;j < 5;j++) {
            kernel[i][j] /= kernelSum;
        }
    }

    printf("Gaussian blur kernel:\n");
    for (int i = 0;i < 5;i++) {
        for (int j = 0;j < 5;j++) {
            printf("%f ", kernel[i][j]);
        }
        printf("\n");
    }

    int kernelDataSize = 5 * 5 * sizeof(float);
    float* deviceKernel;
    hipMalloc(&deviceKernel, kernelDataSize);
    hipMemcpy(deviceKernel, (float*)kernel, kernelDataSize, hipMemcpyHostToDevice);

    unsigned char* deviceImageDataCopy;
    hipMalloc(&deviceImageDataCopy, imageDataSize);
    hipMemcpy(deviceImageDataCopy, deviceImageData, imageDataSize, hipMemcpyDeviceToDevice);

    gaussianBlur << <gridSize, blockSize >> > (deviceKernel, deviceImageDataCopy, deviceImageData, width, height);

    hipFree(deviceKernel);

    printf("Applying Sobel operator\n");
    float* deviceMagnitudes;
    hipMalloc(&deviceMagnitudes, imageDataSize * sizeof(float));
    generateMagnitudes << <gridSize, blockSize >> > (deviceImageData, deviceMagnitudes, width, height);

    lowThreshold *= 255;
    highThreshold *= 255;

    hipMemcpy(deviceImageDataCopy, deviceImageData, imageDataSize, hipMemcpyDeviceToDevice);
    sobel << <gridSize, blockSize >> > (deviceImageDataCopy, deviceImageData, deviceMagnitudes, width, height, lowThreshold, highThreshold);

    hipFree(deviceMagnitudes);

    hipMemcpy(deviceImageDataCopy, deviceImageData, imageDataSize, hipMemcpyDeviceToDevice);
    hysteresis << <gridSize, blockSize >> > (deviceImageDataCopy, deviceImageData, width, height);

    hipFree(deviceImageDataCopy);

    unsigned char* hostImageData = (unsigned char*)malloc(imageDataSize);
    hipMemcpy(hostImageData, deviceImageData, imageDataSize, hipMemcpyDeviceToHost);

    hipFree(deviceImageData);

    cv::Mat modifiedImage = cv::Mat(height, width, CV_8UC3, hostImageData);
    cv::namedWindow("Canny edge detection", cv::WINDOW_AUTOSIZE);
    cv::imshow("Canny edge detection", modifiedImage);
    cv::waitKey(0);

    free(hostImageData);
    return 0;
}
